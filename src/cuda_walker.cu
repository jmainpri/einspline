#include "hip/hip_runtime.h"
#include "cuda_walker.h"
#include "determinant_update.h"

cuda_determinant::cuda_determinant() : 
  N(0), A(NULL), Ainv(NULL), Ainv_delta(NULL), Ainv_colk(0),
  new_row(NULL), delta(0)
{

};

cuda_determinant::cuda_determinant(int n)
{
  resize(N);
}

void
cuda_determinant::resize(int n)
{
  N = n;
  hipMalloc((void**)&A         , N*N*sizeof(float));
  hipMalloc((void**)&Ainv      , N*N*sizeof(float));
  hipMalloc((void**)&Ainv_delta, 1*N*sizeof(float));
  hipMalloc((void**)&Ainv_colk , 1*N*sizeof(float));
  hipMalloc((void**)&new_row   , 1*N*sizeof(float));
  hipMalloc((void**)&delta     , 1*N*sizeof(float));
}

void
cuda_walker::resize(int nup, int ndown) 
{
  N[0] = nup; N[1] = ndown;
  dets[0].resize(N[0]);
  dets[1].resize(N[1]);
}



cuda_population::cuda_population() : MaxPop(1000)
{
  A_vec.resize(MaxPop);
  Ainv_vec.resize(MaxPop);
  delta_vec.resize(MaxPop);
  Ainv_delta_vec.resize(MaxPop);
  Ainv_colk_vec.resize(MaxPop);
  ratio_vec.resize(MaxPop);
  pos_vec.resize(3*MaxPop);


  hipMalloc((void**) &A_list_d,          MaxPop*sizeof(float*));
  hipMalloc((void**) &Ainv_list_d,       MaxPop*sizeof(float*));
  hipMalloc((void**) &Ainv_delta_list_d, MaxPop*sizeof(float*));
  hipMalloc((void**) &Ainv_colk_list_d,  MaxPop*sizeof(float*));
  hipMalloc((void**) &delta_list_d,      MaxPop*sizeof(float*));
  hipMalloc((void**) &ratios_d,          MaxPop*sizeof(float));
  hipMalloc((void**) &pos_d,           4*MaxPop*sizeof(float));
}


__global__ static void
update_inverse_cuda1 (float *Ainv_g[], float *u_g[], float *Ainv_u_g[],
		      float *Ainv_colk_g[], int N, int rowstride, int k);
__global__ static void
update_inverse_cuda2 (float *Ainv_g[], float *u_g[], float *Ainv_u_g[],
		      float *Ainv_colk_g[], int N, int rowstride, int k);


void
cuda_population::calc_new_row(int elec)
{
  int detnum = (elec < num_elecs[0]) ? 0 : 1;
  int N = num_elecs[detnum];
  for (int wi=0; wi<walkers.size(); wi++) {
    cuda_walker &w = walkers[wi];
    cuda_determinant &det = w.dets[detnum];
    pos_vec[4*wi+0] = w.R[3*elec+0];
    pos_vec[4*wi+1] = w.R[3*elec+1];
    pos_vec[4*wi+2] = w.R[3*elec+2];
    delta_vec[wi] = det.delta;
  }
  hipMemcpy(pos_d, &(pos_vec[0]), 4*walkers.size()*sizeof(float), 
	     hipMemcpyHostToDevice);
  hipMemcpy(delta_list_d, &(delta_vec[0]), walkers.size()*sizeof(float*), 
	     hipMemcpyHostToDevice);

  dim3 dimBlock(SPLINE_BLOCK_SIZE);
  dim3 dimGrid (N/SPLINE_BLOCK_SIZE, walkers.size());
  
  eval_multi_multi_UBspline_3d_s_cuda<<<dimGrid,dimBlock>>>
    (pos_d, multi_spline->gridInv, multi_spline->coefs,
     delta_list_d, multi_spline->stride);

}


void 
cuda_population::update_determinants(int elec)
{
  int index=0;
  int detnum = (elec < num_elecs[0]) ? 0 : 1;
  int N = num_elecs[detnum];
  int row = (elec < num_elecs[0]) ? elec : elec - num_elecs[0];
  for (int wi=0; wi<walkers.size(); wi++) {
    cuda_walker &w = walkers[wi];
    cuda_determinant &det = w.dets[detnum];
    if (w.accept) {
      Ainv_vec[index]       = det.Ainv;
      Ainv_delta_vec[index] = det.Ainv_delta;
      Ainv_colk_vec[index]  = det.Ainv_colk;
      delta_vec[index]      = det.delta;
      index++;
    }
  }
  int num_accept = index;

  hipMemcpy (Ainv_list_d, &(Ainv_vec[0]), 
	      num_accept*sizeof(float*), hipMemcpyHostToDevice);
  hipMemcpy (Ainv_delta_list_d, &(Ainv_delta_vec[0]),
	      num_accept*sizeof(float*), hipMemcpyHostToDevice);
  hipMemcpy (Ainv_colk_list_d, &(Ainv_colk_vec[0]), 
	      num_accept*sizeof(float*), hipMemcpyHostToDevice);
  hipMemcpy (delta_list_d, &(delta_vec[0]), 
	      num_accept*sizeof(float*), hipMemcpyHostToDevice);

  dim3 dimBlock(DET_BLOCK_SIZE);
  dim3 dimGrid (N/DET_BLOCK_SIZE, num_accept);
  
  update_inverse_cuda1<<<dimGrid,dimBlock>>>
      (Ainv_list_d, delta_list_d, Ainv_delta_list_d, 
       Ainv_colk_list_d, N, N, row);
  update_inverse_cuda2<<<dimGrid,dimBlock>>>
      (Ainv_list_d, delta_list_d, Ainv_delta_list_d, 
       Ainv_colk_list_d, N, N, row);
};
