#include "hip/hip_runtime.h"
#define BLOCK_SIZE 64

#include <stdio.h>

__global__ void 
eval_multi_UBspline_3d_cuda_c (float *coefs, float *abc, float *vals,
			       int ix, int iy, int iz,
			       int xs, int ys, int zs, int N)
{
  int block = blockIdx.x;
  int thr   = threadIdx.x;
  int offset = block*BLOCK_SIZE+thr;
  __shared__ float abcs[64];
  abcs[thr] = abc[thr];
  

  float val= 0.0;
  //int index=0;
  val = 0.0;
  for (int i=0; i<4; i++)
    for (int j=0; j<4; j++)
      for (int k=0; k<4; k++) {
	float *base_addr = coefs + (ix+i)*xs + (iy+j)*ys + (iz+k)*zs;
	//val += abc[(16*i+4*j+k)*BLOCK_SIZE + thr] * base_addr[offset];
	val += abcs[16*i+4*j+k] * base_addr[offset];	
	//index++;
      }
  vals[offset] = val;
}


__global__ void 
eval_multi_UBspline_3d_cuda_c (float3 r,
                               float *coefs, float *vals,
			       int xs, int ys, int zs, int N)
{
  int block = blockIdx.x;
  int thr   = threadIdx.x;

  __shared__ float abcs[64];
  abcs[thr] = abc[thr];

  float dxInv = 0.0625f;
  float v, dv;

  v = floor(dxInv*r.x);
  dv = dxInv*r.x - v;
  int ix = (int) v;

  v = floor(dxInv*r.x);
  dv = dxInv*r.x - v;
  int iy = (int) v;

  v = floor(dxInv*r.y);
  dv = dxInv*r.y - v;
  int iz = (int) v;




  int ix = (int) v;
  int iy = (int)floor(r.y);
  int iz = (int)floor(r.z);

  int offset = block*BLOCK_SIZE+thr;
  __shared__ float abcs[64];
  abcs[thr] = abc[thr];
  

  float val= 0.0;
  //int index=0;
  val = 0.0;
  for (int i=0; i<4; i++)
    for (int j=0; j<4; j++)
      for (int k=0; k<4; k++) {
	float *base_addr = coefs + (ix+i)*xs + (iy+j)*ys + (iz+k)*zs;
	//val += abc[(16*i+4*j+k)*BLOCK_SIZE + thr] * base_addr[offset];
	val += abcs[16*i+4*j+k] * base_addr[offset];	
	//index++;
      }
  vals[offset] = val;
}


void
test_cuda()
{
  float *coefs  , *abc  , *abc2, *vals;
  float *coefs_d, *abc_d, *vals_d;
  int xs, ys, zs, N;
  int Nx, Ny, Nz;

  N = 4096;
  Nx = Ny = Nz = 16;
  xs = Nx*Ny*Nz;
  ys = Ny*Nz;
  zs = Nz;
  
  int size = Nx*Ny*Nz*N*sizeof(float);
  posix_memalign((void**)&coefs, 16, size);
  hipMalloc((void**)&coefs_d, size);
  for (int ix=0; ix<Nx; ix++)
    for (int iy=0; iy<Ny; iy++)
      for (int iz=0; iz<Nz; iz++)
	for (int n=0; n<N; n++)
	  coefs[ix*xs + iy*ys + iz*zs + n] = drand48();
  hipMemcpy(coefs_d, coefs, size, hipMemcpyHostToDevice);

  posix_memalign ((void**)&abc, 16, 64*sizeof(float));
  posix_memalign ((void**)&abc2, 16, 64*BLOCK_SIZE*sizeof(float));
  hipMalloc((void**)&abc_d, 64*BLOCK_SIZE*sizeof(float));
  for (int i=0; i<64; i++) {
    abc[i] = drand48();
    for (int j=0; j<BLOCK_SIZE; j++)
      abc2[i*BLOCK_SIZE+j] = abc[i];
  }
  //  hipMemcpy(abc_d, abc2, 64*BLOCK_SIZE*sizeof(float), 
  //     hipMemcpyHostToDevice);
  hipMemcpy(abc_d, abc, 64*sizeof(float), 
	     hipMemcpyHostToDevice);

  posix_memalign((void**)&vals, 16, N*sizeof(float));
  hipMalloc((void**)&vals_d, N*sizeof(float));

  dim3 dimBlock(BLOCK_SIZE);
  dim3 dimGrid(N/BLOCK_SIZE);

  int ix=1; 
  int iy=2;
  int iz=3;
  
  clock_t start, end;
  start = clock();
  for (int i=0; i<100000; i++) {
    eval_multi_UBspline_3d_cuda_c<<<dimGrid,dimBlock>>> 
      (coefs_d, abc_d, vals_d, ix, iy, iz, xs, ys, zs, N);
  }
  end = clock();
  double time = (double)(end-start)/(double)(CLOCKS_PER_SEC*100000*N);
  fprintf (stderr, "Evals per second = %1.8e\n", 1.0/time);

  hipMemcpy (vals, vals_d, N*sizeof(float), hipMemcpyDeviceToHost);

  float vals2[N];
  
  for (int n=0; n<N; n++) {
    vals2[n] = 0.0;
    int index=0;
    for(int i=0; i<4; i++)
      for (int j=0; j<4; j++)
	for (int k=0; k<4; k++)  {
	  vals2[n] += abc[index] * coefs[(ix+i)*xs+(iy+j)*ys+(iz+k)*zs+n];
	  index++;
	}
  }


  for (int i=0; i<N/256; i++)	
    fprintf (stderr, "%1.9f %1.9f\n", vals[i], vals2[i]); 


  hipFree(abc_d);
  hipFree(coefs_d);
  hipFree(vals_d);
}


main()
{
  test_cuda();
}
