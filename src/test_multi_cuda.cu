#include "hip/hip_runtime.h"
#include "multi_bspline.h"
#include "multi_bspline_create_cuda.h"
#include "multi_bspline_structs_cuda.h"
#include "multi_bspline_eval_cuda.h"


void
test_float()
{
  int numWalkers = 1000;
  float *vals[numWalkers], *grads[numWalkers], *hess[numWalkers];
  float *coefs, __device__ **vals_d, **grads_d, **hess_d;
  float *r_d, *r_h;
  int xs, ys, zs, N;
  int Nx, Ny, Nz;

  N = 128;
  Nx = Ny = Nz = 32;
  xs = Ny*Nz*N;
  ys = Nz*N;
  zs = N;

  // Setup Bspline coefficients
  int size = Nx*Ny*Nz*N*sizeof(float);
  posix_memalign((void**)&coefs, 16, size);
  for (int ix=0; ix<Nx; ix++)
    for (int iy=0; iy<Ny; iy++)
      for (int iz=0; iz<Nz; iz++)
	for (int n=0; n<N; n++)
	  coefs[ix*xs + iy*ys + iz*zs + n] = drand48();

  Ugrid x_grid, y_grid, z_grid;
  x_grid.start = 0.0; x_grid.end = 1.0; x_grid.num = Nx;
  y_grid.start = 0.0; y_grid.end = 1.0; y_grid.num = Ny;
  z_grid.start = 0.0; z_grid.end = 1.0; z_grid.num = Nz;
  BCtype_s xBC, yBC, zBC;
  xBC.lCode = xBC.rCode = PERIODIC;
  yBC.lCode = yBC.rCode = PERIODIC;
  zBC.lCode = zBC.rCode = PERIODIC;
  

  multi_UBspline_3d_s *spline = 
    create_multi_UBspline_3d_s (x_grid, y_grid, z_grid, xBC, yBC, zBC, N);
  for (int i=0; i<N; i++) 
    set_multi_UBspline_3d_s (spline, i, coefs);

  multi_UBspline_3d_s_cuda *cudaspline = 
    create_multi_UBspline_3d_s_cuda (spline);

  // Setup device value storage
  int numVals = N*numWalkers*10;
  float *valBlock_d, *valBlock_h;
  hipMalloc((void**)&(valBlock_d),     numVals*sizeof(float));
  hipHostMalloc((void**)&(valBlock_h), numVals*sizeof(float));
  hipMalloc((void**)&(vals_d),  numWalkers*sizeof(float*));
  hipMalloc((void**)&(grads_d), numWalkers*sizeof(float*));
  hipMalloc((void**)&(hess_d),  numWalkers*sizeof(float*));
  fprintf (stderr, "valBlock_d = %p\n", valBlock_d);
  for (int i=0; i<numWalkers; i++) {
    vals[i]  = valBlock_d + i*N;
    grads[i] = valBlock_d + N*numWalkers + 3*i*N;
    hess[i]  = valBlock_d + 4*N*numWalkers + 6*i*N;
  }
  hipMemcpy(vals_d,  vals,  numWalkers*sizeof(float*), hipMemcpyHostToDevice);
  hipMemcpy(grads_d, grads, numWalkers*sizeof(float*), hipMemcpyHostToDevice);
  hipMemcpy(hess_d,  hess,  numWalkers*sizeof(float*), hipMemcpyHostToDevice);
  fprintf (stderr, "Finished cuda allocations.\n");

  // Setup walker positions
  hipMalloc((void**)&(r_d),     3*numWalkers*sizeof(float));
  hipHostMalloc((void**)&(r_h), 3*numWalkers*sizeof(float));

  for (int ir=0; ir<numWalkers; ir++) {
    r_h[3*ir+0] = 0.5*drand48();
    r_h[3*ir+1] = 0.5*drand48();
    r_h[3*ir+2] = 0.5*drand48();
  }

  dim3 dimBlock(SPLINE_BLOCK_SIZE);
  dim3 dimGrid(N/SPLINE_BLOCK_SIZE,numWalkers);
  
  float vals_host[N], vals_cuda[N];

  // Check value
  for (int w=0; w<numWalkers; w++) {
    eval_multi_UBspline_3d_s (spline, r_h[3*w+0], r_h[3*w+1], r_h[3*w+2], vals_host);
    hipMemcpy(r_d, r_h, 3*numWalkers*sizeof(float), hipMemcpyHostToDevice);
    eval_multi_multi_UBspline_3d_s_cuda (cudaspline, r_d, vals_d, numWalkers);
    hipMemcpy(vals_cuda, valBlock_d+(N*w), N*sizeof(float), hipMemcpyDeviceToHost);
    //for (int i=0; i<N; i++)
      fprintf (stderr, "%3i  %15.8e %15.8e\n", w, vals_host[0], vals_cuda[0]);
  }


  clock_t start, end;
  start = clock();
  for (int i=0; i<10000; i++) {
    if ((i%1000) == 0) 
      fprintf (stderr, "i = %d\n", i);
    hipMemcpy(r_d, r_h, 3*numWalkers*sizeof(float), hipMemcpyHostToDevice);
    eval_multi_multi_UBspline_3d_s_cuda (cudaspline, r_d, vals_d, numWalkers);
  }
  end = clock();
  double time = (double)(end-start)/(double)((double)CLOCKS_PER_SEC*(double)10000*N*numWalkers);
  fprintf (stderr, "Evals per second = %1.8e\n", 1.0/time);

  start = clock();
  for (int i=0; i<10000; i++) {
    if ((i%1000) == 0) 
      fprintf (stderr, "i = %d\n", i);
    hipMemcpy(r_d, r_h, 3*numWalkers*sizeof(float), hipMemcpyHostToDevice);
    eval_multi_multi_UBspline_3d_s_vgh_cuda (cudaspline, r_d, vals_d, grads_d, hess_d, numWalkers);
  }
  end = clock();
  time = (double)(end-start)/(double)((double)CLOCKS_PER_SEC*(double)10000*N*numWalkers);
  fprintf (stderr, "VGH Evals per second = %1.8e\n", 1.0/time);
  
  hipFree (spline->coefs);
  hipFree (valBlock_d);
  hipFree (vals_d);
  hipFree (grads_d);
  hipFree (hess_d);
  hipFree (r_d);
}



void
test_double()
{
  int numWalkers = 1000;
  double *vals[numWalkers], *grads[numWalkers], *hess[numWalkers];
  double *coefs, __device__ **vals_d, **grads_d, **hess_d;
  double *r_d, *r_h;
  int xs, ys, zs, N;
  int Nx, Ny, Nz;

  N = 128;
  Nx = Ny = Nz = 32;
  xs = Ny*Nz*N;
  ys = Nz*N;
  zs = N;

  // Setup Bspline coefficients
  int size = Nx*Ny*Nz*N*sizeof(double);
  posix_memalign((void**)&coefs, 16, size);
  for (int ix=0; ix<Nx; ix++)
    for (int iy=0; iy<Ny; iy++)
      for (int iz=0; iz<Nz; iz++)
	for (int n=0; n<N; n++)
	  coefs[ix*xs + iy*ys + iz*zs + n] = drand48();

  Ugrid x_grid, y_grid, z_grid;
  x_grid.start = 0.0; x_grid.end = 1.0; x_grid.num = Nx;
  y_grid.start = 0.0; y_grid.end = 1.0; y_grid.num = Ny;
  z_grid.start = 0.0; z_grid.end = 1.0; z_grid.num = Nz;
  BCtype_d xBC, yBC, zBC;
  xBC.lCode = xBC.rCode = PERIODIC;
  yBC.lCode = yBC.rCode = PERIODIC;
  zBC.lCode = zBC.rCode = PERIODIC;
  

  multi_UBspline_3d_d *spline = 
    create_multi_UBspline_3d_d (x_grid, y_grid, z_grid, xBC, yBC, zBC, N);
  for (int i=0; i<N; i++) 
    set_multi_UBspline_3d_d (spline, i, coefs);

  multi_UBspline_3d_d_cuda *cudaspline = 
    create_multi_UBspline_3d_d_cuda (spline);

  // Setup device value storage
  int numVals = N*numWalkers*10;
  double *valBlock_d, *valBlock_h;
  hipMalloc((void**)&(valBlock_d),     numVals*sizeof(double));
  hipHostMalloc((void**)&(valBlock_h), numVals*sizeof(double));
  hipMalloc((void**)&(vals_d),  numWalkers*sizeof(double*));
  hipMalloc((void**)&(grads_d), numWalkers*sizeof(double*));
  hipMalloc((void**)&(hess_d),  numWalkers*sizeof(double*));
  fprintf (stderr, "valBlock_d = %p\n", valBlock_d);
  for (int i=0; i<numWalkers; i++) {
    vals[i]  = valBlock_d + i*N;
    grads[i] = valBlock_d + N*numWalkers + 3*i*N;
    hess[i]  = valBlock_d + 4*N*numWalkers + 6*i*N;
  }
  hipMemcpy(vals_d,  vals,  numWalkers*sizeof(double*), hipMemcpyHostToDevice);
  hipMemcpy(grads_d, grads, numWalkers*sizeof(double*), hipMemcpyHostToDevice);
  hipMemcpy(hess_d,  hess,  numWalkers*sizeof(double*), hipMemcpyHostToDevice);
  fprintf (stderr, "Finished cuda allocations.\n");

  // Setup walker positions
  hipMalloc((void**)&(r_d),     3*numWalkers*sizeof(double));
  hipHostMalloc((void**)&(r_h), 3*numWalkers*sizeof(double));

  for (int ir=0; ir<numWalkers; ir++) {
    r_h[3*ir+0] = 0.5*drand48();
    r_h[3*ir+1] = 0.5*drand48();
    r_h[3*ir+2] = 0.5*drand48();
  }

  dim3 dimBlock(SPLINE_BLOCK_SIZE);
  dim3 dimGrid(N/SPLINE_BLOCK_SIZE,numWalkers);
  
  double vals_host[N], vals_cuda[N];

  // Check value
  for (int w=0; w<numWalkers; w++) {
    eval_multi_UBspline_3d_d (spline, r_h[3*w+0], r_h[3*w+1], r_h[3*w+2], vals_host);
    hipMemcpy(r_d, r_h, 3*numWalkers*sizeof(double), hipMemcpyHostToDevice);
    eval_multi_multi_UBspline_3d_d_cuda (cudaspline, r_d, vals_d, numWalkers);
    hipMemcpy(vals_cuda, valBlock_d+(N*w), N*sizeof(double), hipMemcpyDeviceToHost);
    //for (int i=0; i<N; i++)
      fprintf (stderr, "%3i  %15.8e %15.8e\n", w, vals_host[0], vals_cuda[0]);
  }


  clock_t start, end;
  start = clock();
  for (int i=0; i<10000; i++) {
    if ((i%1000) == 0) 
      fprintf (stderr, "i = %d\n", i);
    hipMemcpy(r_d, r_h, 3*numWalkers*sizeof(double), hipMemcpyHostToDevice);
    eval_multi_multi_UBspline_3d_d_cuda (cudaspline, r_d, vals_d, numWalkers);
  }
  end = clock();
  double time = (double)(end-start)/(double)((double)CLOCKS_PER_SEC*(double)10000*N*numWalkers);
  fprintf (stderr, "Evals per second = %1.8e\n", 1.0/time);

  start = clock();
  for (int i=0; i<10000; i++) {
    if ((i%1000) == 0) 
      fprintf (stderr, "i = %d\n", i);
    hipMemcpy(r_d, r_h, 3*numWalkers*sizeof(double), hipMemcpyHostToDevice);
    eval_multi_multi_UBspline_3d_d_vgh_cuda (cudaspline, r_d, vals_d, grads_d, hess_d, numWalkers);
  }
  end = clock();
  time = (double)(end-start)/(double)((double)CLOCKS_PER_SEC*(double)10000*N*numWalkers);
  fprintf (stderr, "VGH Evals per second = %1.8e\n", 1.0/time);
  
  hipFree (spline->coefs);
  hipFree (valBlock_d);
  hipFree (vals_d);
  hipFree (grads_d);
  hipFree (hess_d);
  hipFree (r_d);
}


main()
{
  fprintf(stderr, "Testing single-precision routines:\n");
  test_float();
  fprintf(stderr, "Testing double-precision routines:\n");
  test_double();
}
