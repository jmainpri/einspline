#include "hip/hip_runtime.h"
#include "multi_bspline.h"
#include "multi_bspline_structs_cuda.h"

__constant__ float Acuda[48];

#include "multi_bspline_cuda_s_impl.h"
#include "multi_bspline_cuda_c.cu"

// typedef struct
// {
//   float *coefs;
//   uint3 stride;
//   float3 gridInv;
//   int num_splines;
// } multi_UBspline_3d_s_cuda;

// typedef struct
// {
//   float *coefs_real, *coefs_imag;
//   uint3 stride;
//   float3 gridInv;
//   int num_splines;
// } multi_UBspline_3d_c_cuda;


extern "C" multi_UBspline_3d_c_cuda*
create_multi_UBspline_3d_c_cuda (multi_UBspline_3d_c* spline)
{
  float A_h[48] = { -1.0/6.0,  3.0/6.0, -3.0/6.0, 1.0/6.0,
		     3.0/6.0, -6.0/6.0,  0.0/6.0, 4.0/6.0,
		    -3.0/6.0,  3.0/6.0,  3.0/6.0, 1.0/6.0,
		     1.0/6.0,  0.0/6.0,  0.0/6.0, 0.0/6.0,
		         0.0,     -0.5,      1.0,    -0.5,
		         0.0,      1.5,     -2.0,     0.0,
		         0.0,     -1.5,      1.0,     0.5,
		         0.0,      0.5,      0.0,     0.0,
		         0.0,      0.0,     -1.0,     1.0,
		         0.0,      0.0,      3.0,    -2.0,
		         0.0,      0.0,     -3.0,     1.0,
		         0.0,      0.0,      1.0,     0.0 };

  hipMemcpyToSymbol(HIP_SYMBOL(Acuda), A_h, 48*sizeof(float), 0, hipMemcpyHostToDevice);

  multi_UBspline_3d_c_cuda *cuda_spline =
    (multi_UBspline_3d_c_cuda*) malloc (sizeof (multi_UBspline_3d_c_cuda*));
  
  cuda_spline->num_splines = spline->num_splines;

  int Nx = spline->x_grid.num+3;
  int Ny = spline->y_grid.num+3;
  int Nz = spline->z_grid.num+3;

  int N = spline->num_splines;
  if ((N%SPLINE_BLOCK_SIZE) != 0)
    N += 64 - (N%SPLINE_BLOCK_SIZE);
  cuda_spline->stride.x = Ny*Nz*N;
  cuda_spline->stride.y = Nz*N;
  cuda_spline->stride.z = N;

  size_t size = Nx*Ny*Nz*N*sizeof(float);

  hipMalloc((void**)&(cuda_spline->coefs_real), size);
  hipMalloc((void**)&(cuda_spline->coefs_imag), size);
  
  float *spline_buff = (float*)malloc(size);

  for (int ix=0; ix<Nx; ix++)
    for (int iy=0; iy<Ny; iy++)
      for (int iz=0; iz<Nz; iz++) 
	for (int isp=0; isp<spline->num_splines; isp++) {
	  spline_buff[ix*cuda_spline->stride.x +
		      iy*cuda_spline->stride.y +
		      iz*cuda_spline->stride.z + isp] =
	    spline->coefs[ix*spline->x_stride +
			  iy*spline->y_stride +
			  iz*spline->z_stride + isp].real();
	}
  hipMemcpy(cuda_spline->coefs_real, spline_buff, size, hipMemcpyHostToDevice);

  for (int ix=0; ix<Nx; ix++)
    for (int iy=0; iy<Ny; iy++)
      for (int iz=0; iz<Nz; iz++) 
	for (int isp=0; isp<spline->num_splines; isp++) {
	  spline_buff[ix*cuda_spline->stride.x +
		      iy*cuda_spline->stride.y +
		      iz*cuda_spline->stride.z + isp] =
	    spline->coefs[ix*spline->x_stride +
			  iy*spline->y_stride +
			  iz*spline->z_stride + isp].imag();
	}
  hipMemcpy(cuda_spline->coefs_imag, spline_buff, size, hipMemcpyHostToDevice);

  free(spline_buff);

  return cuda_spline;
}


extern "C" multi_UBspline_3d_c_cuda*
create_multi_UBspline_3d_c_cuda_conv (multi_UBspline_3d_z* spline)
{
  float A_h[48] = { -1.0/6.0,  3.0/6.0, -3.0/6.0, 1.0/6.0,
		     3.0/6.0, -6.0/6.0,  0.0/6.0, 4.0/6.0,
		    -3.0/6.0,  3.0/6.0,  3.0/6.0, 1.0/6.0,
		     1.0/6.0,  0.0/6.0,  0.0/6.0, 0.0/6.0,
		         0.0,     -0.5,      1.0,    -0.5,
		         0.0,      1.5,     -2.0,     0.0,
		         0.0,     -1.5,      1.0,     0.5,
		         0.0,      0.5,      0.0,     0.0,
		         0.0,      0.0,     -1.0,     1.0,
		         0.0,      0.0,      3.0,    -2.0,
		         0.0,      0.0,     -3.0,     1.0,
		         0.0,      0.0,      1.0,     0.0 };

  hipMemcpyToSymbol(HIP_SYMBOL(Acuda), A_h, 48*sizeof(float), 0, hipMemcpyHostToDevice);

  multi_UBspline_3d_c_cuda *cuda_spline =
    (multi_UBspline_3d_c_cuda*) malloc (sizeof (multi_UBspline_3d_c_cuda*));
  
  cuda_spline->num_splines = spline->num_splines;

  int Nx = spline->x_grid.num+3;
  int Ny = spline->y_grid.num+3;
  int Nz = spline->z_grid.num+3;

  int N = spline->num_splines;
  if ((N%SPLINE_BLOCK_SIZE) != 0)
    N += 64 - (N%SPLINE_BLOCK_SIZE);
  cuda_spline->stride.x = Ny*Nz*N;
  cuda_spline->stride.y = Nz*N;
  cuda_spline->stride.z = N;

  size_t size = Nx*Ny*Nz*N*sizeof(float);

  hipMalloc((void**)&(cuda_spline->coefs_real), size);
  hipMalloc((void**)&(cuda_spline->coefs_imag), size);
  
  float *spline_buff = (float*)malloc(size);

  for (int ix=0; ix<Nx; ix++)
    for (int iy=0; iy<Ny; iy++)
      for (int iz=0; iz<Nz; iz++) 
	for (int isp=0; isp<spline->num_splines; isp++) {
	  spline_buff[ix*cuda_spline->stride.x +
		      iy*cuda_spline->stride.y +
		      iz*cuda_spline->stride.z + isp] =
	    (float)spline->coefs[ix*spline->x_stride +
			  iy*spline->y_stride +
			  iz*spline->z_stride + isp].real();
	}
  hipMemcpy(cuda_spline->coefs_real, spline_buff, size, hipMemcpyHostToDevice);

  for (int ix=0; ix<Nx; ix++)
    for (int iy=0; iy<Ny; iy++)
      for (int iz=0; iz<Nz; iz++) 
	for (int isp=0; isp<spline->num_splines; isp++) {
	  spline_buff[ix*cuda_spline->stride.x +
		      iy*cuda_spline->stride.y +
		      iz*cuda_spline->stride.z + isp] =
	    (float)spline->coefs[ix*spline->x_stride +
			  iy*spline->y_stride +
			  iz*spline->z_stride + isp].imag();
	}
  hipMemcpy(cuda_spline->coefs_imag, spline_buff, size, hipMemcpyHostToDevice);

  free(spline_buff);

  return cuda_spline;
}




extern "C" multi_UBspline_3d_s_cuda*
create_multi_UBspline_3d_s_cuda (multi_UBspline_3d_s* spline)
{
  float A_h[48] = { -1.0/6.0,  3.0/6.0, -3.0/6.0, 1.0/6.0,
		     3.0/6.0, -6.0/6.0,  0.0/6.0, 4.0/6.0,
		    -3.0/6.0,  3.0/6.0,  3.0/6.0, 1.0/6.0,
		     1.0/6.0,  0.0/6.0,  0.0/6.0, 0.0/6.0,
		         0.0,     -0.5,      1.0,    -0.5,
		         0.0,      1.5,     -2.0,     0.0,
		         0.0,     -1.5,      1.0,     0.5,
		         0.0,      0.5,      0.0,     0.0,
		         0.0,      0.0,     -1.0,     1.0,
		         0.0,      0.0,      3.0,    -2.0,
		         0.0,      0.0,     -3.0,     1.0,
		         0.0,      0.0,      1.0,     0.0 };

  hipMemcpyToSymbol(HIP_SYMBOL(Acuda), A_h, 48*sizeof(float), 0, hipMemcpyHostToDevice);

  multi_UBspline_3d_s_cuda *cuda_spline =
    (multi_UBspline_3d_s_cuda*) malloc (sizeof (multi_UBspline_3d_s_cuda*));
  
  cuda_spline->num_splines = spline->num_splines;

  int Nx = spline->x_grid.num+3;
  int Ny = spline->y_grid.num+3;
  int Nz = spline->z_grid.num+3;

  int N = spline->num_splines;
  if ((N%SPLINE_BLOCK_SIZE) != 0)
    N += 64 - (N%SPLINE_BLOCK_SIZE);
  cuda_spline->stride.x = Ny*Nz*N;
  cuda_spline->stride.y = Nz*N;
  cuda_spline->stride.z = N;

  cuda_spline->gridInv.x = spline->x_grid.delta_inv;
  cuda_spline->gridInv.y = spline->y_grid.delta_inv;
  cuda_spline->gridInv.z = spline->z_grid.delta_inv;

  size_t size = Nx*Ny*Nz*N*sizeof(float);

  hipMalloc((void**)&(cuda_spline->coefs), size);
  
  float *spline_buff = (float*)malloc(size);

  for (int ix=0; ix<Nx; ix++)
    for (int iy=0; iy<Ny; iy++)
      for (int iz=0; iz<Nz; iz++) 
	for (int isp=0; isp<spline->num_splines; isp++) {
	  spline_buff[ix*cuda_spline->stride.x +
		      iy*cuda_spline->stride.y +
		      iz*cuda_spline->stride.z + isp] =
	    spline->coefs[ix*spline->x_stride +
			  iy*spline->y_stride +
			  iz*spline->z_stride + isp];
	}
  hipMemcpy(cuda_spline->coefs, spline_buff, size, hipMemcpyHostToDevice);

  free(spline_buff);

  return cuda_spline;
}



extern "C" multi_UBspline_3d_s_cuda*
create_multi_UBspline_3d_s_cuda_conv (multi_UBspline_3d_d* spline)
{
  float A_h[48] = { -1.0/6.0,  3.0/6.0, -3.0/6.0, 1.0/6.0,
		     3.0/6.0, -6.0/6.0,  0.0/6.0, 4.0/6.0,
		    -3.0/6.0,  3.0/6.0,  3.0/6.0, 1.0/6.0,
		     1.0/6.0,  0.0/6.0,  0.0/6.0, 0.0/6.0,
		         0.0,     -0.5,      1.0,    -0.5,
		         0.0,      1.5,     -2.0,     0.0,
		         0.0,     -1.5,      1.0,     0.5,
		         0.0,      0.5,      0.0,     0.0,
		         0.0,      0.0,     -1.0,     1.0,
		         0.0,      0.0,      3.0,    -2.0,
		         0.0,      0.0,     -3.0,     1.0,
		         0.0,      0.0,      1.0,     0.0 };

  hipMemcpyToSymbol(HIP_SYMBOL(Acuda), A_h, 48*sizeof(float), 0, hipMemcpyHostToDevice);

  multi_UBspline_3d_s_cuda *cuda_spline =
    (multi_UBspline_3d_s_cuda*) malloc (sizeof (multi_UBspline_3d_s_cuda*));
  
  cuda_spline->num_splines = spline->num_splines;

  int Nx = spline->x_grid.num+3;
  int Ny = spline->y_grid.num+3;
  int Nz = spline->z_grid.num+3;

  int N = spline->num_splines;
  if ((N%SPLINE_BLOCK_SIZE) != 0)
    N += 64 - (N%SPLINE_BLOCK_SIZE);
  cuda_spline->stride.x = Ny*Nz*N;
  cuda_spline->stride.y = Nz*N;
  cuda_spline->stride.z = N;

  cuda_spline->gridInv.x = spline->x_grid.delta_inv;
  cuda_spline->gridInv.y = spline->y_grid.delta_inv;
  cuda_spline->gridInv.z = spline->z_grid.delta_inv;

  size_t size = Nx*Ny*Nz*N*sizeof(float);

  hipMalloc((void**)&(cuda_spline->coefs), size);
  
  float *spline_buff = (float*)malloc(size);

  for (int ix=0; ix<Nx; ix++)
    for (int iy=0; iy<Ny; iy++)
      for (int iz=0; iz<Nz; iz++) 
	for (int isp=0; isp<spline->num_splines; isp++) {
	  spline_buff[ix*cuda_spline->stride.x +
		      iy*cuda_spline->stride.y +
		      iz*cuda_spline->stride.z + isp] =
	    spline->coefs[ix*spline->x_stride +
			  iy*spline->y_stride +
			  iz*spline->z_stride + isp];
	}
  hipMemcpy(cuda_spline->coefs, spline_buff, size, hipMemcpyHostToDevice);

  free(spline_buff);

  return cuda_spline;
}




